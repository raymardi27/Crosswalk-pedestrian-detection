#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>

__global__ void blurKernel(int left, int top, int right, int bottom, uchar3* input, uchar3* output, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x + left;
    int row = blockIdx.y * blockDim.y + threadIdx.y + top;

    if (col < left || col >= right || row < top || row >= bottom) return;

    int count = 0;
    float sumB = 0, sumG = 0, sumR = 0;

    for (int i = -1; i <= 1; i++) {
        for (int j = -1; j <= 1; j++) {
            int currentX = col + j;
            int currentY = row + i;
            if (currentX >= left && currentX < right && currentY >= top && currentY < bottom) {
                uchar3 pixel = input[currentY * width + currentX];
                sumB += pixel.x;
                sumG += pixel.y;
                sumR += pixel.z;
                count++;
            }
        }
    }

    uchar3& outPixel = output[row * width + col];
    outPixel.x = static_cast<unsigned char>(sumB / count);
    outPixel.y = static_cast<unsigned char>(sumG / count);
    outPixel.z = static_cast<unsigned char>(sumR / count);
}

void cudaBlur(cv::Mat& frame, int left, int top, int right, int bottom) {
    // Ensure correct dimensions
    left = std::max(0, left);
    right = std::min(frame.cols, right);
    top = std::max(0, top);
    bottom = std::min(frame.rows, bottom);

    // Image dimensions
    const int width = frame.cols;
    const int height = frame.rows;

    size_t bytes = width * height * sizeof(uchar3);
    uchar3 *d_input, *d_output;

    // Allocate device memory
    hipMalloc(&d_input, bytes);
    hipMalloc(&d_output, bytes);

    // Copy data to device
    hipMemcpy(d_input, frame.data, bytes, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 numBlocks((right - left + blockSize.x - 1) / blockSize.x, (bottom - top + blockSize.y - 1) / blockSize.y);

    blurKernel<<<numBlocks, blockSize>>>(left, top, right, bottom, d_input, d_output, width, height);

    // Copy result back to host
    hipMemcpy(frame.data, d_output, bytes, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}
